#include "hip/hip_runtime.h"
//                  _  _
//  _   _|_ _  _|o_|__|_
// (_||_||_(_)(_|| |  |
//
// automatic differentiation made easier in C++
// https://github.com/autodiff/autodiff
//
// Licensed under the MIT License <http://opensource.org/licenses/MIT>.
//
// Copyright © 2018–2024 Allan Leal
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// autodiff includes
#include <autodiff/forward/real.hpp>
#include <tests/utils/catch.hpp>
using namespace autodiff;

namespace {
template<typename To, typename Callable>
__global__ void assignKernel(std::size_t n, To* to, Callable callee)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }
    to[tid] = callee();
}

template<typename To, typename Callable>
void assign(To& to, Callable&& callee)
{
    To* toDevicePtr;
    CHECK(hipMalloc(&toDevicePtr, sizeof(To)) == hipSuccess);
    assignKernel<To, Callable><<<1, 1>>>(1, toDevicePtr, callee);
    CHECK(hipMemcpy(&to, toDevicePtr, sizeof(To), hipMemcpyDeviceToHost) == hipSuccess);
    CHECK(hipFree(toDevicePtr) == hipSuccess);
}

template<typename From, typename To, typename Callable>
__global__ void unaryKernel(const From* from, std::size_t n, To* to, Callable callee)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }
    to[tid] = callee(from[tid]);
}

template<typename From, typename To, typename Callable>
void unary(const From& from, To& to, Callable&& callee)
{
    From* fromDevicePtr;
    To* toDevicePtr;
    CHECK(hipMalloc(&fromDevicePtr, sizeof(From)) == hipSuccess);
    CHECK(hipMalloc(&toDevicePtr, sizeof(To)) == hipSuccess);
    CHECK(hipMemcpy(fromDevicePtr, &from, sizeof(From), hipMemcpyHostToDevice) == hipSuccess);
    unaryKernel<From, To, Callable><<<1, 1>>>(fromDevicePtr, 1, toDevicePtr, callee);
    CHECK(hipMemcpy(&to, toDevicePtr, sizeof(To), hipMemcpyDeviceToHost) == hipSuccess);
    CHECK(hipFree(toDevicePtr) == hipSuccess);
    CHECK(hipFree(fromDevicePtr) == hipSuccess);
}

template<typename FromA, typename FromB, typename To, typename Callable>
__global__ void binaryKernel(const FromA* fromA, std::size_t n, const FromB* fromB, To* to, Callable callee)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }
    to[tid] = callee(fromA[tid], fromB[tid]);
}

template<typename FromA, typename FromB, typename To, typename Callable>
void binary(const FromA& fromA, const FromB& fromB, To& to, Callable&& callee)
{
    FromA* fromADevicePtr;
    FromB* fromBDevicePtr;
    To* toDevicePtr;
    CHECK(hipMalloc(&fromADevicePtr, sizeof(FromA)) == hipSuccess);
    CHECK(hipMalloc(&fromBDevicePtr, sizeof(FromB)) == hipSuccess);
    CHECK(hipMalloc(&toDevicePtr, sizeof(To)) == hipSuccess);
    CHECK(hipMemcpy(fromADevicePtr, &fromA, sizeof(FromA), hipMemcpyHostToDevice) == hipSuccess);
    CHECK(hipMemcpy(fromBDevicePtr, &fromB, sizeof(FromB), hipMemcpyHostToDevice) == hipSuccess);
    binaryKernel<FromA, FromB, To, Callable><<<1, 1>>>(fromADevicePtr, 1, fromBDevicePtr, toDevicePtr, callee);
    CHECK(hipMemcpy(&to, toDevicePtr, sizeof(To), hipMemcpyDeviceToHost) == hipSuccess);
    CHECK(hipFree(toDevicePtr) == hipSuccess);
    CHECK(hipFree(fromBDevicePtr) == hipSuccess);
    CHECK(hipFree(fromADevicePtr) == hipSuccess);
}

template<typename FromA, typename FromB, typename FromC, typename To, typename Callable>
__global__ void ternaryKernel(const FromA* fromA, std::size_t n, const FromB* fromB, const FromC* fromC, To* to, Callable callee)
{
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }
    to[tid] = callee(fromA[tid], fromB[tid], fromC[tid]);
}

template<typename FromA, typename FromB, typename FromC, typename To, typename Callable>
void ternary(const FromA& fromA, const FromB& fromB, const FromC& fromC, To& to, Callable&& callee)
{
    FromA* fromADevicePtr;
    FromB* fromBDevicePtr;
    FromC* fromCDevicePtr;
    To* toDevicePtr;
    CHECK(hipMalloc(&fromADevicePtr, sizeof(FromA)) == hipSuccess);
    CHECK(hipMalloc(&fromBDevicePtr, sizeof(FromB)) == hipSuccess);
    CHECK(hipMalloc(&fromCDevicePtr, sizeof(FromC)) == hipSuccess);
    CHECK(hipMalloc(&toDevicePtr, sizeof(To)) == hipSuccess);
    CHECK(hipMemcpy(fromADevicePtr, &fromA, sizeof(FromA), hipMemcpyHostToDevice) == hipSuccess);
    CHECK(hipMemcpy(fromBDevicePtr, &fromB, sizeof(FromB), hipMemcpyHostToDevice) == hipSuccess);
    CHECK(hipMemcpy(fromCDevicePtr, &fromC, sizeof(FromC), hipMemcpyHostToDevice) == hipSuccess);
    ternaryKernel<FromA, FromB, FromC, To, Callable><<<1, 1>>>(fromADevicePtr, 1, fromBDevicePtr, fromCDevicePtr, toDevicePtr, callee);
    CHECK(hipMemcpy(&to, toDevicePtr, sizeof(To), hipMemcpyDeviceToHost) == hipSuccess);
    CHECK(hipFree(toDevicePtr) == hipSuccess);
    CHECK(hipFree(fromCDevicePtr) == hipSuccess);
    CHECK(hipFree(fromBDevicePtr) == hipSuccess);
    CHECK(hipFree(fromADevicePtr) == hipSuccess);
}
} // namespace

#define CHECK_4TH_ORDER_REAL_NUMBERS(a, b) \
    CHECK_APPROX(a[0], b[0]);              \
    CHECK_APPROX(a[1], b[1]);              \
    CHECK_APPROX(a[2], b[2]);              \
    CHECK_APPROX(a[3], b[3]);              \
    CHECK_APPROX(a[4], b[4]);

#define CHECK_DERIVATIVES_REAL4TH_WRT(expr)                                                          \
{                                                                                                    \
    real4th x = 5, y = 7;                                                                            \
    auto f = [] __host__ __device__(const real4th& x, const real4th& y) -> real4th { return expr; }; \
    /* Check directional derivatives of f(x,y) along direction (3, 5) */                             \
    decltype(derivatives(f, along(3, 5), at(x, y))) dfdv;                                            \
                                                                                                     \
    binary(x, y, dfdv, [=] __device__(real4th x, real4th y) {                                        \
        auto dfdv = derivatives(f, along(3, 5), at(x, y));                                           \
        return dfdv;                                                                                 \
    });                                                                                              \
    x[1] = 3.0;                                                                                      \
    y[1] = 5.0;                                                                                      \
    u = expr;                                                                                        \
    x[1] = 0.0;                                                                                      \
    y[1] = 0.0;                                                                                      \
    CHECK_APPROX(dfdv[0], u[0]);                                                                     \
    CHECK_APPROX(dfdv[1], u[1]);                                                                     \
    CHECK_APPROX(dfdv[2], u[2]);                                                                     \
    CHECK_APPROX(dfdv[3], u[3]);                                                                     \
    CHECK_APPROX(dfdv[4], u[4]);                                                                     \
}

// Auxiliary constants
#define ln10 (2.302585092994046)
#define pi (3.14159265359)

TEST_CASE("testing autodiff::real", "[forward][real]")
{
    real4th x, y, z, u, v, w;

    unary(x, x, [] __host__ __device__(real4th x) -> real4th { x = 1.0; return x; });

    CHECK_APPROX(x[0], 1.0);
    CHECK_APPROX(x[1], 0.0);
    CHECK_APPROX(x[2], 0.0);
    CHECK_APPROX(x[3], 0.0);
    CHECK_APPROX(x[4], 0.0);

    unary(x, x, [] __host__ __device__(real4th x) -> real4th { x = {0.5, 3.0, -5.0, -15.0, 11.0}; return x; });

    CHECK_APPROX(x[0], 0.5);
    CHECK_APPROX(x[1], 3.0);
    CHECK_APPROX(x[2], -5.0);
    CHECK_APPROX(x[3], -15.0);
    CHECK_APPROX(x[4], 11.0);

    y = +x;
    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return +x; });

    CHECK_APPROX(y[0], x[0]);
    CHECK_APPROX(y[1], x[1]);
    CHECK_APPROX(y[2], x[2]);
    CHECK_APPROX(y[3], x[3]);
    CHECK_APPROX(y[4], x[4]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return -x; });

    CHECK_APPROX(y[0], -x[0]);
    CHECK_APPROX(y[1], -x[1]);
    CHECK_APPROX(y[2], -x[2]);
    CHECK_APPROX(y[3], -x[3]);
    CHECK_APPROX(y[4], -x[4]);

    z = x + y;
    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return x + y; });

    CHECK_APPROX(z[0], x[0] + y[0]);
    CHECK_APPROX(z[1], x[1] + y[1]);
    CHECK_APPROX(z[2], x[2] + y[2]);
    CHECK_APPROX(z[3], x[3] + y[3]);
    CHECK_APPROX(z[4], x[4] + y[4]);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return x + 1.0; });

    CHECK_APPROX(z[0], x[0] + 1.0);
    CHECK_APPROX(z[1], x[1]);
    CHECK_APPROX(z[2], x[2]);
    CHECK_APPROX(z[3], x[3]);
    CHECK_APPROX(z[4], x[4]);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return 1.0 + x; });

    CHECK_APPROX(z[0], x[0] + 1.0);
    CHECK_APPROX(z[1], x[1]);
    CHECK_APPROX(z[2], x[2]);
    CHECK_APPROX(z[3], x[3]);
    CHECK_APPROX(z[4], x[4]);

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return x - y; });

    CHECK_APPROX(z[0], x[0] - y[0]);
    CHECK_APPROX(z[1], x[1] - y[1]);
    CHECK_APPROX(z[2], x[2] - y[2]);
    CHECK_APPROX(z[3], x[3] - y[3]);
    CHECK_APPROX(z[4], x[4] - y[4]);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return x - 1.0; });

    CHECK_APPROX(z[0], x[0] - 1.0);
    CHECK_APPROX(z[1], x[1]);
    CHECK_APPROX(z[2], x[2]);
    CHECK_APPROX(z[3], x[3]);
    CHECK_APPROX(z[4], x[4]);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return 1.0 - x; });

    CHECK_APPROX(z[0], 1.0 - x[0]);
    CHECK_APPROX(z[1], -x[1]);
    CHECK_APPROX(z[2], -x[2]);
    CHECK_APPROX(z[3], -x[3]);
    CHECK_APPROX(z[4], -x[4]);

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return x * y; });

    CHECK_APPROX(z[0], x[0] * y[0]);
    CHECK_APPROX(z[1], x[1] * y[0] + x[0] * y[1]);
    CHECK_APPROX(z[2], x[2] * y[0] + 2 * x[1] * y[1] + x[0] * y[2]);
    CHECK_APPROX(z[3], x[3] * y[0] + 3 * x[2] * y[1] + 3 * x[1] * y[2] + x[0] * y[3]);
    CHECK_APPROX(z[4], x[4] * y[0] + 4 * x[3] * y[1] + 6 * x[2] * y[2] + 4 * x[1] * y[3] + x[0] * y[4]);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return x * 3.0; });

    CHECK_APPROX(z[0], x[0] * 3.0);
    CHECK_APPROX(z[1], x[1] * 3.0);
    CHECK_APPROX(z[2], x[2] * 3.0);
    CHECK_APPROX(z[3], x[3] * 3.0);
    CHECK_APPROX(z[4], x[4] * 3.0);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return 5.0 * x; });

    CHECK_APPROX(z[0], 5.0 * x[0]);
    CHECK_APPROX(z[1], 5.0 * x[1]);
    CHECK_APPROX(z[2], 5.0 * x[2]);
    CHECK_APPROX(z[3], 5.0 * x[3]);
    CHECK_APPROX(z[4], 5.0 * x[4]);

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return x / y; });

    CHECK_APPROX(z[0], (x[0]) / y[0]);
    CHECK_APPROX(z[1], (x[1] - y[1] * z[0]) / y[0]);
    CHECK_APPROX(z[2], (x[2] - y[2] * z[0] - 2 * y[1] * z[1]) / y[0]);
    CHECK_APPROX(z[3], (x[3] - y[3] * z[0] - 3 * y[2] * z[1] - 3 * y[1] * z[2]) / y[0]);
    CHECK_APPROX(z[4], (x[4] - y[4] * z[0] - 4 * y[3] * z[1] - 6 * y[2] * z[2] - 4 * y[1] * z[3]) / y[0]);

    unary(y, z, [] __host__ __device__(real4th y) -> real4th { return 3.0 / y; });

    CHECK_APPROX(z[0], 3.0 / y[0]);
    CHECK_APPROX(z[1], -(y[1] * z[0]) / y[0]);
    CHECK_APPROX(z[2], -(y[2] * z[0] + 2 * y[1] * z[1]) / y[0]);
    CHECK_APPROX(z[3], -(y[3] * z[0] + 3 * y[2] * z[1] + 3 * y[1] * z[2]) / y[0]);
    CHECK_APPROX(z[4], -(y[4] * z[0] + 4 * y[3] * z[1] + 6 * y[2] * z[2] + 4 * y[1] * z[3]) / y[0]);

    unary(y, z, [] __host__ __device__(real4th y) -> real4th { return y / 5.0; });

    CHECK_APPROX(z[0], y[0] / 5.0);
    CHECK_APPROX(z[1], y[1] / 5.0);
    CHECK_APPROX(z[2], y[2] / 5.0);
    CHECK_APPROX(z[3], y[3] / 5.0);
    CHECK_APPROX(z[4], y[4] / 5.0);

    //=====================================================================================================================
    //
    // TESTING EXPONENTIAL AND LOGARITHMIC FUNCTIONS
    //
    //=====================================================================================================================

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return exp(x); });

    CHECK_APPROX(y[0], exp(x[0]));
    CHECK_APPROX(y[1], x[1] * y[0]);
    CHECK_APPROX(y[2], x[2] * y[0] + x[1] * y[1]);
    CHECK_APPROX(y[3], x[3] * y[0] + 2 * x[2] * y[1] + x[1] * y[2]);
    CHECK_APPROX(y[4], x[4] * y[0] + 3 * x[3] * y[1] + 3 * x[2] * y[2] + x[1] * y[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return log(x); });

    CHECK_APPROX(y[0], log(x[0]));
    CHECK_APPROX(y[1], (x[1]) / x[0]);
    CHECK_APPROX(y[2], (x[2] - x[1] * y[1]) / x[0]);
    CHECK_APPROX(y[3], (x[3] - x[2] * y[1] - 2 * x[1] * y[2]) / x[0]);
    CHECK_APPROX(y[4], (x[4] - x[3] * y[1] - 3 * x[2] * y[2] - 3 * x[1] * y[3]) / x[0]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return log10(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return log(x) / ln10; });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return sqrt(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return exp(0.5 * log(x)); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return cbrt(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return exp(1.0 / 3.0 * log(x)); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return pow(x, x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return exp(x * log(x)); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return pow(x, pi); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return exp(pi * log(x)); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return pow(pi, x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return exp(x * log(pi)); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    //=====================================================================================================================
    //
    // TESTING TRIGONOMETRIC FUNCTIONS
    //
    //=====================================================================================================================

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return sin(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return cos(x); });

    CHECK_APPROX(y[0], sin(x[0]));
    CHECK_APPROX(z[0], cos(x[0]));
    CHECK_APPROX(y[1], x[1] * z[0]);
    CHECK_APPROX(z[1], -x[1] * y[0]);
    CHECK_APPROX(y[2], x[2] * z[0] + x[1] * z[1]);
    CHECK_APPROX(z[2], -x[2] * y[0] - x[1] * y[1]);
    CHECK_APPROX(y[3], x[3] * z[0] + 2 * x[2] * z[1] + x[1] * z[2]);
    CHECK_APPROX(z[3], -x[3] * y[0] - 2 * x[2] * y[1] - x[1] * y[2]);
    CHECK_APPROX(y[4], x[4] * z[0] + 3 * x[3] * z[1] + 3 * x[2] * z[2] + x[1] * z[3]);
    CHECK_APPROX(z[4], -x[4] * y[0] - 3 * x[3] * y[1] - 3 * x[2] * y[2] - x[1] * y[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return tan(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return sin(x) / cos(x); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    //=====================================================================================================================
    //
    // TESTING INVERSE TRIGONOMETRIC FUNCTIONS
    //
    //=====================================================================================================================

    real4th xprime = {{x[1], x[2], x[3], x[4]}};

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return asin(x); });
    binary(x, xprime, z, [] __host__ __device__(real4th x, real4th xprime) -> real4th { return xprime / sqrt(1 - x * x); });

    CHECK_APPROX(y[0], asin(x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return acos(x); });
    binary(x, xprime, z, [] __host__ __device__(real4th x, real4th xprime) -> real4th { return -xprime / sqrt(1 - x * x); });

    CHECK_APPROX(y[0], acos(x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return atan(x); });
    binary(x, xprime, z, [] __host__ __device__(real4th x, real4th xprime) -> real4th { return xprime / (1 + x * x); });

    CHECK_APPROX(y[0], atan(x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    // atan2(double, real4th)
    constexpr double c = 2.0;
    unary(x, y, [=] __host__ __device__(real4th x) -> real4th { return atan2(c, x); });
    binary(x, xprime, z, [=] __host__ __device__(real4th x, real4th xprime) -> real4th { return xprime * (-c / (c * c + x * x)); });

    CHECK_APPROX(y[0], atan2(c, x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    // atan2(real4th, double)
    unary(x, y, [=] __host__ __device__(real4th x) -> real4th { return atan2(x, c); });
    binary(x, xprime, z, [=] __host__ __device__(real4th x, real4th xprime) -> real4th { return xprime * (c / (c * c + x * x)); });

    CHECK_APPROX(y[0], atan2(x[0], c));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    // atan2(real4th, real4th)
    real4th yprime = {{y[1], y[2], y[3], y[4]}};

    real4th s;
    binary(y, x, s, [] __host__ __device__(real4th y, real4th x) -> real4th { real4th s = atan2(y, x); return s; });
    unary(x, z, [=] __host__ __device__(real4th x) -> real4th { return (x[0] * yprime - y[0] * xprime) / (x[0] * x[0] + y[0] * y[0]); });

    CHECK_APPROX(s[0], atan2(y[0], x[0]));
    CHECK_APPROX(s[1], z[0]);
    CHECK_APPROX(s[2], z[1]);
    CHECK_APPROX(s[3], z[2]);
    CHECK_APPROX(s[4], z[3]);

    //=====================================================================================================================
    //
    // TESTING HYPERBOLIC FUNCTIONS
    //
    //=====================================================================================================================
    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return sinh(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return cosh(x); });

    CHECK_APPROX(y[0], sinh(x[0]));
    CHECK_APPROX(z[0], cosh(x[0]));
    CHECK_APPROX(y[1], x[1] * z[0]);
    CHECK_APPROX(z[1], x[1] * y[0]);
    CHECK_APPROX(y[2], x[2] * z[0] + x[1] * z[1]);
    CHECK_APPROX(z[2], x[2] * y[0] + x[1] * y[1]);
    CHECK_APPROX(y[3], x[3] * z[0] + 2 * x[2] * z[1] + x[1] * z[2]);
    CHECK_APPROX(z[3], x[3] * y[0] + 2 * x[2] * y[1] + x[1] * y[2]);
    CHECK_APPROX(y[4], x[4] * z[0] + 3 * x[3] * z[1] + 3 * x[2] * z[2] + x[1] * z[3]);
    CHECK_APPROX(z[4], x[4] * y[0] + 3 * x[3] * y[1] + 3 * x[2] * y[2] + x[1] * y[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return tanh(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return sinh(x) / cosh(x); });

    CHECK_4TH_ORDER_REAL_NUMBERS(y, z);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return asinh(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return 1 / sqrt(x * x + 1); });

    CHECK_APPROX(y[0], asinh(x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return acosh(10*x); /* acosh requires x > 1 */ });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return 1 / sqrt(100 * x * x - 1); });

    CHECK_APPROX(y[0], acosh(10 * x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return atanh(x); });
    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return 1 / (1 - x * x); });

    CHECK_APPROX(y[0], atanh(x[0]));
    CHECK_APPROX(y[1], z[0]);
    CHECK_APPROX(y[2], z[1]);
    CHECK_APPROX(y[3], z[2]);
    CHECK_APPROX(y[4], z[3]);

    //=====================================================================================================================
    //
    // TESTING OTHER FUNCTIONS
    //
    //=====================================================================================================================

    y = abs(x);
    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return abs(x); });

    CHECK_APPROX(y[0], std::abs(x[0]));
    CHECK_APPROX(y[1], std::abs(x[0]) / x[0] * x[1]);
    CHECK_APPROX(y[2], std::abs(x[0]) / x[0] * x[2]);
    CHECK_APPROX(y[3], std::abs(x[0]) / x[0] * x[3]);
    CHECK_APPROX(y[4], std::abs(x[0]) / x[0] * x[4]);

    unary(x, y, [] __host__ __device__(real4th x) -> real4th { return -x; });
    unary(y, z, [] __host__ __device__(real4th y) -> real4th { return abs(y); });

    CHECK_APPROX(z[0], std::abs(y[0]));
    CHECK_APPROX(z[1], std::abs(y[0]) / (y[0]) * y[1]);
    CHECK_APPROX(z[2], std::abs(y[0]) / (y[0]) * y[2]);
    CHECK_APPROX(z[3], std::abs(y[0]) / (y[0]) * y[3]);
    CHECK_APPROX(z[4], std::abs(y[0]) / (y[0]) * y[4]);

    //=====================================================================================================================
    //
    // TESTING MIN/MAX FUNCTIONS
    //
    //=====================================================================================================================

    x = {0.5, 3.0, -5.0, -15.0, 11.0};
    y = {4.5, 3.0, -5.0, -15.0, 11.0};

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return min(x, y); });
    CHECK(z == x);

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return min(y, x); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return min(x, 0.1); });
    CHECK(z == real4th(0.1));

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return min(0.2, x); });
    CHECK(z == real4th(0.2));

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return min(0.5, x); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return min(x, 0.5); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return min(3.5, x); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return min(x, 3.5); });
    CHECK(z == x);

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return max(x, y); });
    CHECK(z == y);

    binary(x, y, z, [] __host__ __device__(real4th x, real4th y) -> real4th { return max(y, x); });
    CHECK(z == y);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return max(x, 0.1); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return max(0.2, x); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return max(0.5, x); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return max(x, 0.5); });
    CHECK(z == x);

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return max(8.5, x); });
    CHECK(z == real4th(8.5));

    unary(x, z, [] __host__ __device__(real4th x) -> real4th { return max(x, 8.5); });
    CHECK(z == real4th(8.5));

    //=====================================================================================================================
    //
    // TESTING COMPARISON OPERATORS
    //
    //=====================================================================================================================

    x = {0.5, 3.0, -5.0, -15.0, 11.0};

    // Check equality not only on value but also on the derivatives
    bool cond;
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x == real4th({0.5, 3.0, -5.0, -15.0, 11.0}); });
    CHECK(cond);

    // Check equality against plain numeric types (double) do not require check against derivatives
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x == 0.6; });
    CHECK_FALSE(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x == 0.5; });
    CHECK(cond);

    // Check inequalities
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x == real4th({0.5, 3.1, -5.0, -15.0, 11.0}); });
    CHECK_FALSE(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x != real4th({0.5, 3.1, -5.0, -15.0, 11.0}); });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x != 1.0; });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x < 1.0; });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x > 0.1; });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x <= 1.0; });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x >= 0.1; });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x <= 0.5; });
    CHECK(cond);
    unary(x, cond, [] __host__ __device__(real4th x) -> bool { return x >= 0.5; });
    CHECK(cond);

    //=====================================================================================================================
    //
    // TESTING DERIVATIVE CALCULATIONS
    //
    //=====================================================================================================================

    CHECK_DERIVATIVES_REAL4TH_WRT(exp(log(2 * x + 3 * y)));
    CHECK_DERIVATIVES_REAL4TH_WRT(sin(2 * x + 3 * y));
    CHECK_DERIVATIVES_REAL4TH_WRT(exp(2 * x + 3 * y) * log(x / y));

    // Testing array-unpacking of derivatives for real number
    {
        real4th x = {{2.0, 3.0, 4.0, 5.0, 6.0}};

        decltype(derivatives(x)) xDerivatives;
        unary(x, xDerivatives, [] __host__ __device__(real4th x) { return derivatives(x); });
        auto [x0, x1, x2, x3, x4] = xDerivatives;

        CHECK_APPROX(x0, x[0]);
        CHECK_APPROX(x1, x[1]);
        CHECK_APPROX(x2, x[2]);
        CHECK_APPROX(x3, x[3]);
        CHECK_APPROX(x4, x[4]);
    }

//    // Testing array-unpacking of derivatives for vector of real numbers
//    {
//        real4th x = {{2.0, 3.0, 4.0, 5.0, 6.0}};
//        real4th y = {{3.0, 4.0, 5.0, 6.0, 7.0}};
//        real4th z = {{4.0, 5.0, 6.0, 7.0, 8.0}};
//
//        std::vector<real4th> u = { x, y, z };
//
//        auto [u0, u1, u2, u3, u4] = derivatives(u);
//
//        CHECK_APPROX( u0[0], x[0] ); CHECK_APPROX( u1[0], x[1] ); CHECK_APPROX( u2[0], x[2] ); CHECK_APPROX( u3[0], x[3] ); CHECK_APPROX( u4[0], x[4] );
//        CHECK_APPROX( u0[1], y[0] ); CHECK_APPROX( u1[1], y[1] ); CHECK_APPROX( u2[1], y[2] ); CHECK_APPROX( u3[1], y[3] ); CHECK_APPROX( u4[1], y[4] );
//        CHECK_APPROX( u0[2], z[0] ); CHECK_APPROX( u1[2], z[1] ); CHECK_APPROX( u2[2], z[2] ); CHECK_APPROX( u3[2], z[3] ); CHECK_APPROX( u4[2], z[4] );
//    }
}
